#include "cudaClaw_main_noVis.h"

#include <iostream>

// Defined falgs to switch between GPUs for debug or run
#define GPU_RELEASE 0
#define GPU_DEBUG 1

int main(int argc, char** argv)
{
	setupCUDA();

	// Boundary setup
	//boundaryConditions<BC_left_reflective, BC_right_reflective, BC_up_reflective, BC_down_reflective> reflective_conditions;
	//boundaryConditions<BC_left_reflective, BC_right_reflective, BC_up_reflective, BC_down_absorbing> semi_reflective_conditions;

	//BC_left_reflective left;
	//BC_right_reflective right;
	//BC_up_reflective up;
	//BC_down_reflective down;

	//reflective_conditions.condition_left = left;
	//reflective_conditions.condition_right = right;
	//reflective_conditions.condition_up = up;
	//reflective_conditions.condition_down = down;

	boundaryConditions<BC_left_absorbing, BC_right_absorbing, BC_up_absorbing, BC_down_absorbing> absorbing_conditions;
	BC_left_absorbing left;
	BC_right_absorbing right;
	BC_up_absorbing up;
	BC_down_absorbing down;

	absorbing_conditions.condition_left = left;
	absorbing_conditions.condition_right = right;
	absorbing_conditions.condition_up = up;
	absorbing_conditions.condition_down = down;

	// Solver setup
	// acoustics
	acoustics_horizontal acoustic_h;
	acoustics_vertical   acoustic_v;
	// shallow water
	shallow_water_horizontal shallow_water_h;
	shallow_water_vertical   shallow_water_v;

	// Limiter setup
	limiter_MC phi;
	limiter_superbee phi1;

	// Entropy fix
	entropy_fix_Shallow_Water_horziontal ent_fix_shallow_water_h;
	entropy_fix_Shallow_Water_vertical   ent_fix_shallow_water_v;
	null_entropy no_entropy;

	int cellsX = 128;
	int cellsY = 128;
	real ratio = (real)cellsY/(real)cellsX;

	real simulation_start_time = 0.0f;
	real simulation_end_time = 1.0f;

	real snapshotRate = 0.00f;
	bool entropy_fix = true;

	pdeParam problemParam = setup(cellsX, cellsY, -1, 1, -1, 1, entropy_fix, simulation_start_time, simulation_end_time, snapshotRate, radial_plateau);
	//real* coeffs = new real[2];
	//coeffs[0] = 4.0f;
	//coeffs[1] = 1.0f;

	//pdeParam problemParam = setup(cellsX, cellsY, 0, 1, 0, 1, entropy_fix, simulation_start_time, simulation_end_time, snapshotRate, centered_circle_q, uniform_coefficients, coeffs);
	//delete coeffs;

	//solvePDE(problemParam, acoustic_h, acoustic_v, phi, reflective_conditions, no_entropy, no_entropy);
	solvePDE(problemParam, shallow_water_h, shallow_water_v, phi, absorbing_conditions, ent_fix_shallow_water_h, ent_fix_shallow_water_v);

	problemParam.clean();
	gracefulExit();
}

template<class Solver_h, class Solver_v, class Limiter, class Conditions, class Entropy_h, class Entropy_v>
void solvePDE(pdeParam &params, Solver_h solver_h, Solver_v solver_v, Limiter phi, Conditions conds, Entropy_h ent_fix_h, Entropy_v ent_fix_v)
{
	// This single step seems necessary for the data to show
	step(params, solver_h, solver_v, phi, conds, ent_fix_h, ent_fix_v);

	if( params.snapshots )
		params.takeSnapshot(0, "pde data");	// take initial state snapshot

	int snap_number = 1;
	
	real simulationTime = 0.0f;
	real simulationStepTime = 0.0f;
	real simulationTimeInterval = 0.0f;
	while (simulationTime < params.endTime)
	{
		simulationStepTime = step(params, solver_h, solver_v, phi, conds, ent_fix_h, ent_fix_v);
		printf("Simulation Time is: %fs\n", simulationTime);

		simulationTime += simulationStepTime;
		simulationTimeInterval += simulationStepTime;
		if (params.snapshots && simulationTimeInterval > params.snapshotTimeInterval)
		{
			params.takeSnapshot(snap_number, "pde data");
			simulationTimeInterval = 0.0f;
			snap_number++;
		}
	}
}

void setupCUDA()
{
	int device = GPU_RELEASE;	//1 for debug 0 for run, chooses the gpu

	hipError_t errorDevice = hipSetDevice(device);

	hipDeviceProp_t device_property;
	hipGetDeviceProperties(&device_property, device);

	// Some error when choosing cache configuration, could be with the order of the call, 
	if (device_property.major < 2)
		// cache-shared memory configuring not possible, no cache
		printf("Cache configuration not possible\n");
	else
	{
		//hipError_t errorCachePref1 = hipFuncSetCacheConfig(reinterpret_cast<const void*>("fused_Riemann_limiter_horizontal_update_kernel"), hipFuncCachePreferShared);
		//hipError_t errorCachePref2 = hipFuncSetCacheConfig(reinterpret_cast<const void*>("fused_Riemann_limiter_vertical_update_kernel"), hipFuncCachePreferShared);
		//printf("Cache configuration done, config1: %i, config2: %i\n",errorCachePref1,errorCachePref2);
	}
}
template <class T>
inline void getCudaAttribute(T *attribute, hipDeviceAttribute_t device_attribute,
							 int device)
{
	// Credit to Nvidia GPU computing SDK, deviceQuery project.
	hipError_t error = hipDeviceGetAttribute(attribute, device_attribute, device);

	if( hipSuccess != error)
	{
		printf("cuSafeCallNoSync() Driver API error = %04d\n", error);
        exit(-1);
    }
}
void gracefulExit()
{
	hipDeviceReset();
	exit(0);
}
