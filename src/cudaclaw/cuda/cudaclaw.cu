#include "cudaclaw.h"
#include "common.h"
#include "problem_setup.h"

#include "boundary_conditions.h"
#include "fused_Riemann_Limiter.h"

#define GPU_RELEASE 0
#define GPU_DEBUG 1

void setupCUDA();

template <class T>
inline void getCudaAttribute(T *attribute, hipDeviceAttribute_t device_attribute);


pdeParam *param;
boundaryConditions<BC_left_absorbing,
		       BC_right_absorbing,
		       BC_up_absorbing,
		       BC_down_absorbing> bc;

shallow_water_horizontal shallow_water_h;
shallow_water_vertical   shallow_water_v;
limiter_MC phi_mc;
real* cpu_q;
size_t qbc_size;
hipError_t err;

// Entropy fix
entropy_fix_Shallow_Water_horziontal entropy_fix_h;
entropy_fix_Shallow_Water_vertical   entropy_fix_v;

int shallow_water_solver_allocate(int cellsX,
				  int cellsY,
				  int ghostCells,
                                  int numStates,
                                  int numWaves,
                                  int numCoeff,
                                  real startX,
                                  real endX,
                                  real startY,
                                  real endY,
                                  real startTime,
                                  real endTime)
{
    param = new pdeParam(cellsX,
			 cellsY,
			 ghostCells,
			 numStates,
			 numWaves,
			 numCoeff,
			 startX,
			 endX,
			 startY,
			 endY,
			 startTime,
			 endTime);

    param->setSnapshotRate(0.0);
    param->setEntropyFix(true);

    qbc_size = param->cellsX*param->cellsY*param->numStates*sizeof(real);

    return 0;
}

int shallow_water_solver_setup (int bc_left,
                                int bc_right,
                                int bc_up,
                                int bc_down,
                                int limiter
                                )
{

    setupCUDA();


    // the boundary conditions and limiter are currently hard-coded to
    // absorbing, mc

    BC_left_absorbing left;
    BC_right_absorbing right;
    BC_up_absorbing up;
    BC_down_absorbing down;

    bc.condition_left = left;
    bc.condition_right = right;
    bc.condition_up = up;
    bc.condition_down = down;

    return 0;
}

int hyperbolic_solver_2d_step (real dt, real* next_dt)
{

    setBoundaryConditions(*param, bc);
    limited_Riemann_Update(*param,
			   shallow_water_h,
			   shallow_water_v,
			   phi_mc,
			   entropy_fix_h,
			   entropy_fix_v);

    err = hipMemcpy(next_dt, param->dt_used, sizeof(real), hipMemcpyDeviceToHost);

    if (err != hipSuccess) {
    	return err;
    }
    return 0;
}

int hyperbolic_solver_2d_get_qbc (real* qbc)
{
    err = hipMemcpy(qbc, param->qNew, qbc_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
    	return err;
    }
    return 0;
}

int hyperbolic_solver_2d_set_qbc (real* qbc)
{
    err = hipMemcpy(param->qNew, qbc, qbc_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
    	return err;
    }
    return 0;
}

void setupCUDA()
{
    int device = GPU_RELEASE;	//1 for debug 0 for run, chooses the gpu

    hipError_t errorDevice = hipSetDevice(device);

    hipDeviceProp_t device_property;
    hipGetDeviceProperties(&device_property, device);

}
template <class T>
inline void getCudaAttribute(T *attribute, hipDeviceAttribute_t device_attribute,
			     int device)
{
    // Credit to Nvidia GPU computing SDK, deviceQuery project.
    hipError_t error = hipDeviceGetAttribute(attribute, device_attribute, device);

    if( hipSuccess != error) {
	printf("cuSafeCallNoSync() Driver API error = %04d\n", error);
	exit(-1);
    }
}
