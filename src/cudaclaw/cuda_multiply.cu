#include <stdio.h>
#include <hip/hip_runtime.h>

/**
 * CUDA Kernel Device code
 *
 * computes A = A*x
 *
 * where A is a vector of length numElements
 */
__global__ void
cudaMultiply(double *A, const double *x, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        A[i] = A[i]*x[0];
    }
}

extern "C" {
void cuda_multiply (double* array, double multiplier, int m, int n) {

    hipError_t err = hipSuccess;

    // Allocate the device vector A
    double *d_A = NULL;

    // Allocate the device multiplier x
    double *d_x = NULL;

    int numElements = m*n;

    size_t size = numElements * sizeof(double);

    err = hipMalloc((void **)&d_A, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    err = hipMalloc((void **)&d_x, sizeof(double));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device multiplier x (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors array and host memory to the 
    // device input vectors in device memory
    printf("Copy input from the numpy array in host memory to the CUDA device\n");
    err = hipMemcpy(d_A, array, size, hipMemcpyHostToDevice);

    printf("Copy multiplier to the CUDA device\n");
    err = hipMemcpy(d_x, &multiplier, sizeof(double), hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy numpy vector from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", 
    	   blocksPerGrid, threadsPerBlock);
    cudaMultiply<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_x, numElements);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch cudaMultiply kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the numpy vector
    // in host memory.
    printf("Copy output data from the CUDA device to the numpy array in host memory\n");
    err = hipMemcpy(array, d_A, size, hipMemcpyDeviceToHost);

    return ;
}
};
